#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \
    if(error != hipSuccess){                                               \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                            \
    }                                                                       \
}                                                                           

void checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0E-8;
    bool match = 1;

    for(int i=0; i<N; i++){
        if(abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            printf("Arrays do not match!]n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if(match){
        printf("Arrays match.\n");
    }

    return;
}

// 初始化数组，生成随机浮点数填充数组ip
void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

// 核函数，在GPU上并行计算两个数组的和，每个线程处理一个元素的加法
__global__ void sumArrayOnGPU(float *A, float *B, float *C, const int N){
    // threadIdx 是一个预定义的变量，类型为 uint3，表示当前线程在块中的索引；
    // 在这个核函数中，每个线程处理一个元素的加法，因此 i 被设置为 threadIdx.x
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N) C[i] = A[i] + B[i];
}

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // 选择CUDA设备
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // 定义数组大小
    int nElem = 1 << 24;
    printf("Vector size %d\n", nElem);

    // 申请CPU内存
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    // 在CPU端初始化时间
    double iStart, iElaps;
    iStart = seconds();
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    iElaps = seconds() - iStart;
    printf("initialData Time elapsed %f sec\n", iElaps);
    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    // CPU端执行数组加法函数
    iStart = seconds();
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    iElaps = seconds() - iStart;
    printf("sumArraysOnHost Time elapsed %f sec\n", iElaps);

    // GPU端申请全局内存
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // 将输入数据从主机CPU传输到设备GPU
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

    int iLen = 1024;
    dim3 block(iLen);
    dim3 grid((nElem+block.x-1)/block.x);

    iStart = seconds();
    sumArrayOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf( "sumArraysOnGPU <<<  %d, %d  >>>  Time elapsed %f sec\n", 
            grid.x, block.x, iElaps);

    // 检查是否有错误信息
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    CHECK(hipDeviceReset());
    return(0);
}

/*
$ nvcc sumArraysOnGPU-timer.cu -o timer
$ ./timer 
./timer Starting...
Using Device 0: NVIDIA GeForce RTX 4070 Laptop GPU
Vector size 16777216
initialData Time elapsed 0.419008 sec
sumArraysOnHost Time elapsed 0.018562 sec
sumArraysOnGPU <<<  32768, 512  >>>  Time elapsed 0.633475 sec
Arrays match.

$ ./timer 
./timer Starting...
Using Device 0: NVIDIA GeForce RTX 4070 Laptop GPU
Vector size 16777216
initialData Time elapsed 0.407757 sec
sumArraysOnHost Time elapsed 0.016320 sec
sumArraysOnGPU <<<  16384, 1024  >>>  Time elapsed 0.000998 sec
Arrays match.
*/