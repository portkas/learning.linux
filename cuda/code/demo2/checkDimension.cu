#include <hip/hip_runtime.h>
#include <stdio.h>

// 在每个线程中打印当前线程的 threadIdx、blockIdx、blockDim 和 gridDim 信息
__global__ void checkIndex(){
    printf( "threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n", 
            threadIdx.x, threadIdx.y, threadIdx.z, 
            blockIdx.x, blockIdx.y, blockIdx.z,
            blockDim.x, blockDim.y, blockDim.z,
            gridDim.x, gridDim.y, gridDim.z);
}

int main(){
    // 定义数据元素总数（程序需要处理的数据元素）
    // 将数据分成多个块，每个块处理一定数量的元素
    int nElem = 6;

    // 定义块的尺寸（一维）
    dim3 block(3);

    // 基于块和数据的大小计算网格尺寸（一维）
    // 每个块可以处理block.x个元素，计算总共需要多少个块来处理所有元素
    // 加block.x-1的目的是为了确保所有数据元素都能被处理，即使数据元素数不是块大小的整数倍
    // 如果 nElem 能被 block.x 整除，那么 nElem / block.x 就是正确的块数
    // 如果 nElem 不能被 block.x 整除，那么 nElem / block.x 会少计算一个块
    // 通过加上 block.x - 1，确保即使有余数，也能正确计算出足够的块数
    dim3 grid( (nElem+block.x-1)/block.x );

    // 主机端程序，用来检查网格和块维度
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
    checkIndex <<<grid, block>>>();
    hipDeviceReset();
    return 0;
}

/*
$ nvcc -arch=sm_80 checkDimension.cu -o check
$ ./check 
grid.x 2 grid.y 1 grid.z 1
block.x 3 block.y 1 block.z 1
threadIdx:(0, 0, 0) blockIdx:(1, 0, 0) blockDim:(3, 1, 1) gridDim:(2, 1, 1)
threadIdx:(1, 0, 0) blockIdx:(1, 0, 0) blockDim:(3, 1, 1) gridDim:(2, 1, 1)
threadIdx:(2, 0, 0) blockIdx:(1, 0, 0) blockDim:(3, 1, 1) gridDim:(2, 1, 1)
threadIdx:(0, 0, 0) blockIdx:(0, 0, 0) blockDim:(3, 1, 1) gridDim:(2, 1, 1)
threadIdx:(1, 0, 0) blockIdx:(0, 0, 0) blockDim:(3, 1, 1) gridDim:(2, 1, 1)
threadIdx:(2, 0, 0) blockIdx:(0, 0, 0) blockDim:(3, 1, 1) gridDim:(2, 1, 1)
*/