#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \
    if (error != hipSuccess) {                                             \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1);                                                            \
    }                                                                       \
}

void initialInt(int *ip, int size){
    for(int i=0; i<size; i++){
        ip[i] = i;
    }
}

void printMatrix(int *C, const int nx, const int ny){
    int *ic = C;
    printf("\nMatrix: (%d.%d)\n", nx, ny);
    for(int iy=0; iy<ny; iy++){
        for(int ix=0; ix<nx; ix++){
            printf("%3d", ic[ix]);
        }
        ic += nx;
        printf("\n");
    }
    printf("\n");
    return;
}

__global__ void printThreadIndex(int *A, const int nx, const int ny){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    printf( "thread_id(%d, %d) block_id(%d, %d) corrdinate(%d, %d) " 
            "global index:%2d ival:%2d\n", threadIdx.x, threadIdx.y, 
            blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // 定义矩阵维度
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // 申请主机内存
    int *h_A;
    h_A = (int *)malloc(nBytes);

    // 主机端初始化矩阵
    for (int i = 0; i < nxy; i++)
    {
        h_A[i] = i;
    }
    printMatrix(h_A, nx, ny);

    // 申请设备端内存
    int *d_MatA;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));

    // 拷贝主机端数据到设备端
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    
    // 设置执行配置
    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1)/block.x, (ny + block.y - 1)/block.y);

    // 执行核函数
    printThreadIndex<<<grid, block>>>(d_MatA, nx, ny);
    CHECK(hipGetLastError());

    // 释放内存
    CHECK(hipFree(d_MatA));
    free(h_A);

    // 释放设备内存
    CHECK(hipDeviceReset());
    return 0;
}