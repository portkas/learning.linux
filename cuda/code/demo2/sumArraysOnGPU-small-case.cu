#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \
    if(error != hipSuccess){                                               \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                            \
    }                                                                       \
}                                                                           

void checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0E-8;
    bool match = 1;

    for(int i=0; i<N; i++){
        if(abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            printf("Arrays do not match!]n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if(match){
        printf("Arrays match.\n");
    }

    return;
}

// 初始化数组，生成随机浮点数填充数组ip
void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

// 核函数，在GPU上并行计算两个数组的和，每个线程处理一个元素的加法
__global__ void sumArrayOnGPU(float *A, float *B, float *C, const int N){
    // threadIdx 是一个预定义的变量，类型为 uint3，表示当前线程在块中的索引；
    // 在这个核函数中，每个线程处理一个元素的加法，因此 i 被设置为 threadIdx.x
    int i = threadIdx.x;
    if(i<N) C[i] = A[i] + B[i];
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // 选择CUDA设备
    int dev = 0;
    CHECK(hipSetDevice(dev));

    // 定义数组大小
    int nElem = 1 << 5;
    printf("Vector size %d\n", nElem);

    size_t nBytes = nElem*sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    // 这里实际上是多余的，因为d_C已经初始化为0
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

    dim3 block(nElem);
    dim3 grid(1);

    sumArrayOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    printf("Execution configure <<<%d, %d>>>\n", grid.x, block.x);

    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    checkResult(hostRef, gpuRef, nElem);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    CHECK(hipDeviceReset());
    return(0);
}

/*
$ nvcc sumArraysOnGPU-small-case.cu -o addvector
$ ./addvector 
./addvector Starting...
Vector size 32
Execution configure <<<1, 32>>>
Arrays match.
*/