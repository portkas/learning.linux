#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \
    if (error != hipSuccess) {                                             \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1);                                                            \
    }                                                                       \
}

int main() {
    int deviceCount;
    CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        printf("There are no available CUDA devices.\n");
    } else {
        printf("There are %d available CUDA devices:\n", deviceCount);
        for (int i = 0; i < deviceCount; i++) {
            hipDeviceProp_t deviceProp;
            CHECK(hipGetDeviceProperties(&deviceProp, i));
            printf("Device %d: %s\n", i, deviceProp.name);
        }
    }

    return 0;
}

/*
$ nvcc cudaDeviceCount.cu -o device
$ ./device 
There are 1 available CUDA devices:
Device 0: NVIDIA GeForce RTX 4070 Laptop GPU
*/