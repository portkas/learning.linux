#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \
    if(error != hipSuccess){                                               \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                            \
    }                                                                       \
} 

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}


// 交错
__global__ void mathKernel1(float *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    if(tid % 2 == 0){
        ia = 100.0f;
    }else{
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

// 按线程束划分
__global__ void mathKernel2(float *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    if((tid / warpSize) % 2 == 0){
        ia = 100.0f;
    }else{
        ib = 200.0f;
    }
    c[tid] = ia + ib;
}

// 和mathKernel1有什么区别吗？？？
__global__ void mathKernel3(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    bool ipred = (tid % 2 == 0);

    if (ipred)
    {
        ia = 100.0f;
    }

    if (!ipred)
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

// 和mathKernel2有什么区别吗？？？
__global__ void mathKernel4(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    int itid = tid >> 5;

    if (itid & 0x01 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

// 和mathKernel2有什么区别吗？？？
__global__ void warmingup(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

    // set up data size
    int size = 64;
    int blocksize = 64;
    if(argc > 1) blocksize = atoi(argv[1]);
    if(argc > 2) size = atoi(argv[2]);
    printf("Data size %d ", size);

    // set up execution configuration
    dim3 block(blocksize, 1);
    dim3 grid((size + block.x -1)/block.x, 1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    // allocate gpu memory
    float *d_C;
    int nBytes = size * sizeof(float);
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // run a warmup kernel to remove overhead
    double iStart, iElaps;
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    warmingup<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("warmup      <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps );
    CHECK(hipGetLastError());

    // run kernel 1
    iStart = seconds();
    mathKernel1<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("mathKernel1 <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps );
    CHECK(hipGetLastError());

    // run kernel 3
    iStart = seconds();
    mathKernel2<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("mathKernel2 <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps );
    CHECK(hipGetLastError());

    // run kernel 3
    iStart = seconds();
    mathKernel3<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("mathKernel3 <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps);
    CHECK(hipGetLastError());

    // run kernel 4
    iStart = seconds();
    mathKernel4<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("mathKernel4 <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps);
    CHECK(hipGetLastError());

    // free gpu memory and reset divece
    CHECK(hipFree(d_C));
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}