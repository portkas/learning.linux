
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__修饰符告诉编译器这个函数将会从GPU中调用，在GPU上执行
__global__ void helloFromGPU(){
    printf("Hello World from GPU!\n");
}

int main(){
    printf("Hello World from CPU!\n");
    // 启动内核函数；
    // 三重尖括号意味着从主线程到设备端代码的调用；
    // 一个内核函数通过一组线程来执行，所有线程执行相同的代码；
    // 有十个线程被调用；
    helloFromGPU <<<1, 10>>>();
    // 显式地释放和清空当前进程中与当前设备有关的所有资源；
    hipDeviceReset();
    return 0;
}

// $ nvcc -arch sm_80 helloFromGPU.cu -o hello
// GeForce RTX 4070 是基于 NVIDIA 的 Ada Lovelace 架构
// -arch sm_80使编译器为Ada lovelace架构生成设备代码；